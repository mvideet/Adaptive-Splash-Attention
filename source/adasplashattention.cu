#include "hip/hip_runtime.h"
#include <cinttypes>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <float.h>




#ifndef BLOCK_M
#define BLOCK_M 32 // number of queries per thread block
#endif
#ifndef BLOCK_N
#define BLOCK_N 64 //number of keys per tile
#endif
#ifndef D_MAX
#define D_MAX 128 // maximum head dimension supported - make itlarger if we have a larger hidden state
#endif
#ifndef K_KEEP
#define K_KEEP 8 //number of top-K attention scores to keep per query and keep it relatively sparse
#endif

#define EPS 1e-6 // to prevent division by 0
#define MAX_HALLEY_ITERS 8 // number of max halley iterations for finding T via gradients

// wraps CUDA calls and automatically checks for errors
#define CUDA_CHECK(call) do { \
    hipError_t err = call; \
    TORCH_CHECK(err == hipSuccess, "CUDA error at ", __FILE__, ":", __LINE__, "-", hipGetErrorString(err)); \
} while(0)

static_assert(BLOCK_N > 0 && BLOCK_M >0, "Block dimensions must be positive");
static_assert(BLOCK_N * D_MAX <=8192, "Shared memory K tile too large"); //each block needs BLOCK_N * D_MAX floats & most GPUs have 48kb shared mmeroy/block
static_assert(BLOCK_M<=1024, "BLOCK_M exceeds max threads per block");
static_assert(K_KEEP <=BLOCK_N, "K_KEEP should not exceed BLOCK_N");
static_assert(D_MAX>=64,"D_MAX should be at least 64 for practical use" );

//inline says that the compiler should inline this function for performacnce
__device__ inline void insert_topk(float val, int idx, float (&vals)[K_KEEP], int (&inds)[K_KEEP]) {
    // vals stores the actual attention scores
    // inds stores which key each score corresponds to (actual pos in the sequence)
    int j = K_KEEP-1;
    while (j>=0 && val > vals[j]) { // compare with current element
        if (j<K_KEEP-1) {
            vals[j+1] = vals[j]; // shift to the right and it discards the last from memory
            inds[j+1] = inds[j];
        }
        --j;
    }
    ++j;
    if (j<K_KEEP) {
        vals[j] = val;
        inds[j] = idx;
    }
}

__device__ void entmax_threshold(const float* s, int k, float alpha, float* p, float &tau, bool full){
const float inv_am1 = 1.f/(alpha-1.f);
//initialize bisection bounds
float lo = (alpha-1.f)*s[k-1] - 1.f; //smallest score
float hi = (alpha-1.f)*s[0]; //largest score

auto eval = [&](float t, float &f, float &fp, float &fpp){
    f = -1;//start at -1 and add each p_i in the loop below
    fp=fpp=0;
    for(int j = 0;j<k;j++){
        float u = (alpha-1.f)*s[j] - t; //compute u_j = (α-1)s_j - τ
        if(u<=0) break;
        float up = powf(u, inv_am1); //u^(1/(α-1)) = probability p_j
        f+=up; //add to constraint sum
        if(full){
            fp += -inv_am1 * powf(u, (2.f-alpha)*inv_am1); //∂f/∂τ
            fpp += inv_am1*(inv_am1+1.f-alpha) * powf(u, (3.f-2*alpha)*inv_am1);
        }
    }
};
tau = 0.5f*(lo+hi); //initialize tau as midpoint
//halley's method iterations
for(int it = 0; it<MAX_HALLEY_ITERS; ++it){
    float f, fp, fpp;
    eval(tau, f, fp, fpp);
    if(fabsf(f)<1e-3f) break;//converged
    float tn = tau; //save current tau
    if(full){//if derivatives are available, then use the updates, else resort to bisection
        float denom = 2.f*fp*fp - f*fpp;
        tn = tau-2.f*f*fp/fmaxf(denom,EPS); //prevent division by 0

    }
    if(!(tn>=lo && tn <=hi)) {
        tn = 0.5f*(lo+hi); //fallback to bisection if new tau is outside bounds
    }
    eval(tn, f, fp, fpp);  // Evaluate function and derivatives at new tau
    if(f>0){
        lo=tn;             // If f>0, tau is too small, update lower bound
    }
    else{
        hi=tn;             // If f<=0, tau is too large, update upper bound  
    }
    tau = tn;             // Update tau for next iteration
}
float norm = 0;
for(int j = 0;j<k;j++){
    float u = (alpha-1.f)*s[j] - tau;   // Compute u_j after tau is optimized
   float pj = (u>0)?powf(u, inv_am1):0;
   p[j] = pj; //store probability
   norm +=pj;
}
norm = fmaxf(norm, EPS);
for(int j = 0;j<k;j++){
    p[j] /= norm;
}
}


__global__ void build_mask_kernel(
    const float* Q, const float* K, const int* Q_idx, const int* K_idx, int B, int H, int NQ, int NK, int d, float alpha, float sm_scale, int8_t* M, float* taus, int nQB, int nKB){
        // Calculate global query index by combining block and thread indices:
        // blockIdx.x gives the block index in x-dimension 
        // BLOCK_M is the thread block size
        // threadIdx.x gives the thread index within the block
        //NQ and NK are the number of queries and keys per head
        //Queries [B*H, NQ,d] input tensor
        //Keys [B*H, NK, d] input tensor
        //Q_idx = query position indices
        //Key_idx = key position indicites [B*H, NK]
        //BLOCK_M is the number of queries processed per thread block
        //remember that there are many threads in a block that can use shared memory
        int q = blockIdx.x*BLOCK_M + threadIdx.x;//global query index
        int bh = blockIdx.y; //which attention head and batch item am i working on
        //each block of threads are tied to one attention head but many queries
        //will multiply across all keys in the batch/head
        if (q>=NQ){
            return;
        }
        int idx = bh*NQ+q;
        int seq_q = Q_idx[idx]; //global seq position of this query
        float q_reg[D_MAX]; //we will store the query vector that is D_head size 
        
        #pragma unroll
        for(int t=0;t<d;++t)
            q_reg[t] = Q[idx*d+t];
        
        float s_top[K_KEEP]; //top-K attention scores and initialize to -infinity
        int ind[K_KEEP]; //indices of top-K elements and initialize to -1
        #pragma unroll
        for(int i=0;i<K_KEEP;++i){
            s_top[i] = -FLT_MAX;
            ind[i] = -1;
        }
        extern __shared__ float shmem[]; //shared memory across the block
        float* Ktile = shmem; //use shared memory for key tile
        for (int start = 0; start<NK; start+=BLOCK_N){ //process keys in tiles of size BLOCK_N
            int tile_id = threadIdx.x; //thread ID within block
            //each thread loads multiple elements
            // We stride by BLOCK_M because that's the number of threads in the block
            // Each thread needs to handle multiple elements (BLOCK_N*d total elements)
            // So we distribute the work evenly across BLOCK_M threads by having each thread
            // process elements spaced BLOCK_M apart
            for(int x = tile_id; x < BLOCK_N*d; x += BLOCK_M){
                // Each thread processes elements at indices: tile_id, tile_id+BLOCK_M, tile_id+2*BLOCK_M, ...
                int col = x/d; //which key in the tile
                int dim = x%d; //which dimension of the key
                int kn = start + col; //global key index - we don't add dim since dim represents the vector dimension (0 to d-1), not the key position. Adding dim would incorrectly offset the key index.
                bool ok = (kn < NK &&                            // Key exists
                          dim < d &&                             // Dimension is valid
                          K_idx[bh*NK + kn] <= seq_q);          // Causal: key position ≤ query position
                
                // Load key element or zero if out of bounds/causal
                Ktile[x] = ok ? K[(bh*NK + kn)*d + dim] : 0.f;  // Load key element if ok, otherwise 0
            // K is a 1D array that represents a 3D tensor [B*H, NK, d]
            // The indexing formula (bh*NK + kn)*d + dim flattens the 3D access:
            // bh*NK*d - moves to the start of the batch+head
            // kn*d - moves to the start of the key vector
            // dim - moves to the specific dimension
            }
        
            __syncthreads(); //wait for all threads to finish loading before proceeding

            for(int j = 0;j<BLOCK_N;j++){
                // Calculate global key index by adding tile offset (start) to local key index (j)
                int kn = start + j;  // start points to beginning of current tile, j is position within tile (0 to BLOCK_N-1)
                
                // Skip if key index is out of bounds (>= NK) or violates causal masking
                // K_idx[bh*NK + kn] gets the sequence position of key kn in batch/head bh:
                //   bh*NK - offset to start of current batch/head's key indices
                //   + kn - offset to specific key's position
                // Compare against seq_q (query's sequence position) for causal masking
                if(kn >= NK || K_idx[bh*NK + kn] > seq_q) continue;
                float sum = 0;
                #pragma unroll
                for(int t=0;t<d;t++)
                    sum+=q_reg[t]*Ktile[j*d+t];
                
                sum *=sm_scale;
                insert_topk(sum, kn, s_top, ind);
            }            
            //compute attention scores for all the keys in the current tile.
            //we have a top-K buffer that stores the top-K attention scores and their corresponding key indices
            //we will use this buffer to compute the attention mask
            //we will use the entmax threshold to compute the attention mask
            //we will use the top-K buffer to compute the attention mask
            //we will use the entmax threshold to compute the attention mask
            __syncthreads(); //wait for all threads to finish before proceeding
        }
        
        float tau;
        float pbuf[K_KEEP]; // Buffer for entmax probabilities
    
        entmax_threshold(s_top, K_KEEP, alpha, pbuf, tau, true); // Compute α-entmax weights
        taus[idx] = tau; // Store threshold for backward pass
        

        int query_block = q/BLOCK_M; //which query block this is
        // Calculate base offset into mask tensor:
        // bh*nQB*nKB - moves to start of current batch+head's mask section
        //   bh - current batch*head index
        //   nQB - number of query blocks
        //   nKB - number of key blocks
        // query_block*nKB - moves to current query block's row
        //   query_block - index of current query block
        //   nKB - number of key blocks (stride between query block rows)
        int base = bh*nQB*nKB + query_block*nKB;
        for(int i = 0;i<K_KEEP;i++){
            if(ind[i] < 0) continue;
            int jKB = ind[i]/BLOCK_N; //which key block contains this key
            M[base + jKB] = 1; //mark this block as needed and we can't remove this from the mask
        }
    }

__global__ void build_lookup_kernel(const int8_t* M, int B, int H, int nQB, int nKB, int* Qi_ptr, int* Qi_idx, int* Kj_ptr, int* Kj_idx){
    int bh = blockIdx.x; //batch*head index from block ID. THis only uses 1D grid since it processes one batch*head per thread
    if(bh>=H*B) return;

    int baseM = bh * nQB * nKB;                              // Base address in mask tensor
    int bQi   = bh * (nQB + 1);  //CSR pointer array always has 1 extra entry (to mark the end):                             // Base address in Qi_ptr
    int bQiI  = bh * (nQB * nKB);   // Base address in Qi_idx array for this batch*head - shape is [B*H, nQB*nKB]
    int bKj   = bh * (nKB + 1);    //CSR pointer always has 1 extra entry to mark the end                         
    int bKjI  = bh * (nQB * nKB);
    Qi_ptr[bQi] = 0; //value is always 0 - shape is [B*H, nQB+1] because it is sparse

    //EG: Query attentnds to [keys] (0, [0, 3]), (1, [1]), (2, [0,2,3])
    //Qi_idx = [0, 3, 1, 0, 2, 3]
    //Qi_ptr = [0, 2, 3,6]
    //Fill in Qi_ptr
    for(int i = 0;i<nQB;i++){
        int c = 0;//active key blocks for this query block counter
        for(int j =0;j<nKB;j++){
            c += M[baseM+i*nKB+j];
        }
        Qi_ptr[bQi+i+1] = Qi_ptr[bQi+i]+c; //It updates the next pointer in the Qi_ptr array by adding the number of active key blocks (c) for the current query block i.

    }
    //Fill in Qi_indx
    for(int i = 0;i<nQB;i++){
        int w = Qi_ptr[bQi+i];  // Starting position in Qi_idx array where we'll store which key blocks this query block attends to
        for(int j =0;j<nKB;j++){
            if(M[baseM+i*nKB+j]){
                // Store the key block index j at position bQiI+w in Qi_idx array
                // bQiI: Base offset for this batch+head's Qi_idx section
                // w: Current write position within this query block's active key indices
                // w++ increments w after storing j, advancing to next write position
                // j: Index of the current key block that is active for this query block
                Qi_idx[bQiI+w++] = j;
            }
    }}
    Kj_ptr[bKj] = 0;//first pointer is always 0
    // Count active query blocks for each key block
    for(int j=0; j<nKB; j++) {
        int c = 0;  // Counter for active query blocks
        for(int i=0; i<nQB; i++) {
            c += M[baseM + i*nKB + j];  // Count if query block i attends to key block j
        }
        Kj_ptr[bKj + j + 1] = Kj_ptr[bKj + j] + c;  // Store cumulative count
    }

    // Fill indices of active query blocks for each key block
    for(int j=0; j<nKB; j++) {
        int w = Kj_ptr[bKj + j];  // Starting write position for this key block
        for(int i=0; i<nQB; i++) {
            if(M[baseM + i*nKB + j]) {  // If query block i attends to key block j
                Kj_idx[bKjI + w++] = i;  // Store query block index
            }
        }
    }
    //


}
